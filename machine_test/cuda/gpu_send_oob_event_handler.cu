#include "hip/hip_runtime.h"
#include <absl/log/check.h>
#include <absl/strings/str_format.h>
#include <errno.h>
#include <fcntl.h>
#include <sys/ioctl.h>

#include "cuda/common.cuh"
#include "machine_test/cuda/gpu_send_oob_event_handler.cuh"
#include "machine_test/include/benchmark_common.h"
#include "machine_test/include/tcpdirect_common.h"

namespace gpudirect_tcpxd {

bool GpuSendOobEventHandler::HandleEvents(unsigned events) {
  if (events & EPOLLOUT) {
    if (!CustomizedHandleEPollOut()) return false;
  }
  if (events & EPOLLERR) {
    if (!HandleEPollErr()) return false;
  }
  return true;
}

bool GpuSendOobEventHandler::CustomizedHandleEPollOut() {
  if (bytes_sent_ == message_size_) {
    if (!PendingSendDone()) {
      return true;
    }
    epoch_tx_bytes_ += message_size_;
    Reset();
  }
  struct cmsghdr *cmsg = CMSG_FIRSTHDR(&msg_);
  cmsg->cmsg_level = SOL_SOCKET;
  cmsg->cmsg_type = SCM_DEVMEM_OFFSET;
  cmsg->cmsg_len = CMSG_LEN(sizeof(int) * 2);
  *((int *)CMSG_DATA(cmsg)) = gpu_page_allocator_->GetGpuMemFd(msg_id_);
  ((int *)CMSG_DATA(cmsg))[1] = (int)message_size_ + 1;

  iov_.iov_base = &(buf_.get())[bytes_sent_];
  iov_.iov_len = message_size_ - bytes_sent_;
  ssize_t ret =
      sendmsg(socket_, &msg_, MSG_ZEROCOPY | MSG_SOCK_DEVMEM | MSG_DONTWAIT);
  if (ret < 0 && errno != EWOULDBLOCK && errno != EAGAIN) {
    error_ =
        absl::StrFormat("sendmsg() error,  ret: %d, errno: %d", ret, errno);
    // PLOG(ERROR) << "sendmsg() error: ";
    return false;
  }
  bytes_sent_ += ret;
  sendmsg_cnt_++;

  if (HasError()) {
    return false;
  }
  return true;
}
}  // namespace gpudirect_tcpxd
