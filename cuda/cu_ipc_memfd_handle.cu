#include "hip/hip_runtime.h"
/*
 Copyright 2024 Google LLC

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

      https://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

#include <absl/log/log.h>
#include <absl/strings/str_format.h>

#include <string>

#include "cuda/common.cuh"
#include "cuda/cu_ipc_memfd_handle.cuh"

namespace gpudirect_tcpxd {

CuIpcMemfdHandle::CuIpcMemfdHandle(int fd, int dev_id, size_t size,
                                   size_t align) {
  LOG(INFO) << absl::StrFormat(
      "Importing CUDA IPC mem from from fd: %ld, dev_id: %ld, size: %ld, "
      "align: %ld",
      fd, dev_id, size, align);
  CU_ASSERT_SUCCESS(hipDeviceGet(&dev_, dev_id));
  CU_ASSERT_SUCCESS(hipDevicePrimaryCtxRetain(&ctx_, dev_));
  size_ = size;
  CU_ASSERT_SUCCESS(
      hipMemImportFromShareableHandle(&handle_, (void*)(long long)fd,
                                     hipMemHandleTypePosixFileDescriptor));
  CU_ASSERT_SUCCESS(hipMemAddressReserve(&ptr_, size_, align, 0, 0));
  CU_ASSERT_SUCCESS(hipMemMap(ptr_, size_, 0, handle_, 0));
  close(fd);
  hipMemAccessDesc desc = {};
  desc.location.type = hipMemLocationTypeDevice;
  desc.location.id = dev_id;
  desc.flags = hipMemAccessFlagsProtReadWrite;
  CU_ASSERT_SUCCESS(hipMemSetAccess(ptr_, size_, &desc, 1 /*count*/));
}

CuIpcMemfdHandle::~CuIpcMemfdHandle() {
  hipMemUnmap(ptr_, size_);
  hipMemRelease(handle_);
  hipMemAddressFree(ptr_, size_);
}

}  // namespace gpudirect_tcpxd
