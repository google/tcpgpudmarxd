#include "hip/hip_runtime.h"
#include "cuda/cu_ipc_memfd_handle.cuh"

#include <string>

#include <absl/log/log.h>
#include "cuda/common.cuh"
#include <absl/strings/str_format.h>

namespace tcpdirect {

CuIpcMemfdHandle::CuIpcMemfdHandle(int fd, int dev_id, size_t size,
                                   size_t align) {
  LOG(INFO) << absl::StrFormat(
      "Importing CUDA IPC mem from from fd: %ld, dev_id: %ld, size: %ld, "
      "align: %ld",
      fd, dev_id, size, align);
  CU_ASSERT_SUCCESS(hipDeviceGet(&dev_, dev_id));
  CU_ASSERT_SUCCESS(hipDevicePrimaryCtxRetain(&ctx_, dev_));
  size_ = size;
  CU_ASSERT_SUCCESS(
      hipMemImportFromShareableHandle(&handle_, (void*)(long long)fd,
                                     hipMemHandleTypePosixFileDescriptor));
  CU_ASSERT_SUCCESS(hipMemAddressReserve(&ptr_, size_, align, 0, 0));
  CU_ASSERT_SUCCESS(hipMemMap(ptr_, size_, 0, handle_, 0));
  close(fd);
  hipMemAccessDesc desc = {};
  desc.location.type = hipMemLocationTypeDevice;
  desc.location.id = dev_id;
  desc.flags = hipMemAccessFlagsProtReadWrite;
  CU_ASSERT_SUCCESS(hipMemSetAccess(ptr_, size_, &desc, 1 /*count*/));
}

CuIpcMemfdHandle::~CuIpcMemfdHandle() {
  hipMemUnmap(ptr_, size_);
  hipMemRelease(handle_);
  hipMemAddressFree(ptr_, size_);
}

}  // namespace tcpdirect
