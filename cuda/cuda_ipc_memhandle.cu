#include "hip/hip_runtime.h"
#include <absl/log/log.h>

#include "cuda/common.cuh"
#include "cuda/cuda_ipc_memhandle.cuh"

namespace gpudirect_tcpxd {

CudaIpcMemhandle::CudaIpcMemhandle(const std::string& handle) {
  memcpy(&mem_handle_, handle.data(), handle.size());
  CU_ASSERT_SUCCESS(
      hipIpcOpenMemHandle(&ptr_, mem_handle_, hipIpcMemLazyEnablePeerAccess));
}

CudaIpcMemhandle::~CudaIpcMemhandle() { hipIpcCloseMemHandle(ptr_); }

}  // namespace gpudirect_tcpxd
