#include "cuda/cuda_context_manager.cuh"

namespace gpudirect_tcpxd {
CudaContextManager::CudaContextManager(int gpu_cuda_idx) {
  CU_ASSERT_SUCCESS(hipDeviceGet(&dev, gpu_cuda_idx));
  CU_ASSERT_SUCCESS(hipCtxCreate(&ctx, 0, dev));
}

CudaContextManager::CudaContextManager(std::string gpu_pci_addr) {
  CU_ASSERT_SUCCESS(hipDeviceGetByPCIBusId(&dev, gpu_pci_addr.c_str()));
  CU_ASSERT_SUCCESS(hipCtxCreate(&ctx, 0, dev));
}

void CudaContextManager::PushContext() {
  CU_ASSERT_SUCCESS(hipCtxPushCurrent(ctx));
}

void CudaContextManager::PopContext() {
  hipCtx_t old_ctx;
  CU_ASSERT_SUCCESS(hipCtxPopCurrent(&old_ctx));
}

CudaContextManager::~CudaContextManager() {
  hipCtx_t old_ctx;
  CU_ASSERT_SUCCESS(hipCtxPopCurrent(&old_ctx));
}
}  // namespace gpudirect_tcpxd
