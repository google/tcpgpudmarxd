#include "hip/hip_runtime.h"
#include <absl/log/check.h>
#include <absl/log/log.h>
#include <absl/status/status.h>

#include <memory>
#include <numeric>
#include <vector>

#include "code.pb.h"
#include "cuda/cuda_context_manager.cuh"
#include "cuda/cuda_ipc_memhandle_exporter.cuh"
#include "cuda/dmabuf_gpu_page_allocator.cuh"
#include "include/unix_socket_server.h"
#include "proto/gpu_rxq_configuration.pb.h"
#include "proto/unix_socket_message.pb.h"

namespace gpudirect_tcpxd {

absl::Status CudaIpcMemhandleExporter::Initialize(
    const GpuRxqConfigurationList& config_list, const std::string& prefix) {
  prefix_ = prefix;
  if (prefix_.back() == '/') {
    prefix_.pop_back();
  }

  telemetry_.Start();

  // Setup CUDA context and DmabufPageAllocator
  LOG(INFO) << "Setting up CUDA context and dmabuf page allocator ...";

  size_t rx_pool_size = RX_POOL_SIZE;

  if (config_list.has_rx_pool_size()) {
    rx_pool_size = config_list.has_rx_pool_size();
  }

  int tcpd_qstart = config_list.rss_set_size();

  for (const auto& gpu_rxq_config : config_list.gpu_rxq_configs()) {
    std::string ifname = gpu_rxq_config.ifname();
    std::string nic_pci_addr = gpu_rxq_config.nic_pci_addr();
    for (const auto& gpu_info : gpu_rxq_config.gpu_infos()) {
      std::string gpu_pci_addr = gpu_info.gpu_pci_addr();
      gpu_pci_binding_map_[gpu_info.gpu_pci_addr()] = {
          .cuda_ctx = std::make_unique<CudaContextManager>(gpu_pci_addr),
          .page_allocator = std::make_unique<DmabufGpuPageAllocator>(
              gpu_pci_addr, nic_pci_addr, /*create_page_pool=*/true,
              rx_pool_size),
          .ifname = ifname,
          .queue_ids = {gpu_info.queue_ids().begin(),
                        gpu_info.queue_ids().end()},
      };
    }
  }

  // 3. Allocate gpu memory, bind rxq, and get cudaIpcMemHandle
  LOG(INFO)
      << "Allocating gpu memory, binding rxq, and getting cudaIpcMemHandle ...";

  for (auto& [gpu_pci, gpu_rxq_binding] : gpu_pci_binding_map_) {
    auto& cuda_ctx = *gpu_rxq_binding.cuda_ctx;
    auto& page_allocator = *gpu_rxq_binding.page_allocator;
    auto& page_id = gpu_rxq_binding.page_id;
    auto& mem_handle = gpu_rxq_binding.mem_handle;
    auto& ifname = gpu_rxq_binding.ifname;
    auto& qids = gpu_rxq_binding.queue_ids;
    cuda_ctx.PushContext();
    bool allocation_success = false;
    page_allocator.AllocatePage(rx_pool_size, &page_id, &allocation_success);

    if (!allocation_success) {
      return absl::UnavailableError("Failed to allocate GPUMEM page: " +
                                    ifname);
    }

    for (int qid : qids) {
      if (int ret =
              gpumem_bind_rxq(page_allocator.GetGpuMemFd(page_id), ifname, qid);
          ret < 0) {
        return absl::UnavailableError("Failed to bind rxq: " + ifname);
      }
    }

    if (auto err = hipIpcGetMemHandle(
            &mem_handle, (void*)page_allocator.GetGpuMem(page_id));
        err != 0) {
      return absl::UnavailableError("Failed to get cudaIpcMemHandle: " +
                                    ifname);
    }

    cuda_ctx.PopContext();
  }
  return absl::OkStatus();
}

absl::Status CudaIpcMemhandleExporter::Export() {
  LOG(INFO) << "Starting Unix socket servers ...";
  // Find memhandle by gpu pci
  us_servers_.emplace_back(std::make_unique<UnixSocketServer>(
      absl::StrFormat("%s/get_gpu_by_gpu_pci", prefix_),
      [this](UnixSocketMessage&& request, UnixSocketMessage* response,
             bool* fin) {
        absl::Time start = absl::Now();
        telemetry_.IncrementRequests();
        UnixSocketProto* mutable_proto = response->mutable_proto();
        std::string* buffer = mutable_proto->mutable_raw_bytes();
        if (!request.has_proto() || !request.proto().has_raw_bytes()) {
          mutable_proto->mutable_status()->set_code(
              google::rpc::Code::INVALID_ARGUMENT);
          mutable_proto->mutable_status()->set_message(
              "Expecting text format request.");
          buffer->append("Error.\n\nExpecting text format request.\n");
          *fin = true;
          telemetry_.IncrementIpcFailure();
          telemetry_.IncrementIpcFailureAndCause(
              mutable_proto->mutable_status()->message());
          return;
        }
        const std::string& gpu_pci = request.proto().raw_bytes();
        GpuRxqBinding& binding = gpu_pci_binding_map_[gpu_pci];
        for (int i = 0; i < sizeof(binding.mem_handle); ++i) {
          buffer->push_back(*((char*)&binding.mem_handle + i));
        }
        telemetry_.IncrementIpcSuccess();
        telemetry_.AddLatency(absl::Now() - start);
      }));

  for (auto& server : us_servers_) {
    if (auto server_status = server->Start(); !server_status.ok()) {
      return server_status;
    }
  }

  LOG(INFO) << "CudaIpcMemHandle Unix socket servers started ...";
  return absl::OkStatus();
}
void CudaIpcMemhandleExporter::Cleanup() {
  for (auto& server : us_servers_) {
    server->Stop();
  }
}
}  // namespace gpudirect_tcpxd