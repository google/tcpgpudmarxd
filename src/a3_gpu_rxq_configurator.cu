#include "hip/hip_runtime.h"
/*
 Copyright 2024 Google LLC

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

      https://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.
 */

#include <absl/flags/flag.h>
#include <absl/log/check.h>
#include <absl/log/log.h>
#include <absl/strings/ascii.h>
#include <dirent.h>
#include <ifaddrs.h>
#include <stdio.h>
#include <sys/types.h>

#include <memory>
#include <vector>

#include "include/a3_gpu_rxq_configurator.cuh"
#include "include/pci_helpers.h"

#define PCI_INFO_LEN 1024
// <2-4 digit domain>:<2-4 digit bus>:<2 digit device>:<1 digit function>
#define MAX_PCI_ADDR_LEN 16
#define MAX_HOPS 4
ABSL_FLAG(
    int, num_hops, 2,
    "Number of hops to the PCIE switch shared by the 2 GPUs and the NIC(s).");

namespace gpudirect_tcpxd {
namespace {
constexpr int kRssSetSize{8};
constexpr int kTcpdQueueCount{8};
}  // namespace

GpuRxqConfigurationList A3GpuRxqConfigurator::GetConfigurations() {
  GpuRxqConfigurationList config_list;
  absl::flat_hash_map<std::string, std::string> netdev_to_pci;
  absl::flat_hash_map<std::string, std::string> pci_to_netdev;
  absl::flat_hash_map<std::string, std::vector<std::string>> netdev_to_gpu_pcis;
  struct ifaddrs *all_ifs = nullptr;
  if (getifaddrs(&all_ifs) != 0 || all_ifs == nullptr) {
    LOG(ERROR) << "Failed to retrieve network ifs, error: " << strerror(errno);
    return config_list;
  }
  struct ifaddrs *head = all_ifs;
  do {
    // Skip non-IPV4 and non-IPV6 interfaces
    if (head->ifa_addr->sa_family != AF_INET &&
        head->ifa_addr->sa_family != AF_INET6) {
      continue;
    }
    // Skip interfaces we have already seen before
    if (netdev_to_pci.contains(head->ifa_name)) continue;
    char if_sysfs_path[PATH_MAX] = {0};
    snprintf(if_sysfs_path, PATH_MAX, "/sys/class/net/%s/device/",
             head->ifa_name);
    char if_sysfs_realpath[PATH_MAX] = {0};
    // Only pick interfaces that has an actual PCI device associated
    if (realpath(if_sysfs_path, if_sysfs_realpath) == nullptr) continue;
    int last_char_idx = strlen(if_sysfs_realpath) - 1;
    if (if_sysfs_realpath[last_char_idx] == '/')
      if_sysfs_realpath[last_char_idx] = '\0';
    int path_length = 0;
    for (int i = 0; i < strlen(if_sysfs_realpath); i++) {
      if (if_sysfs_realpath[i] == '/') ++path_length;
    }
    // The host NIC should be closest to the CPU, exclude it.
    // TODO (penzhao@): consider using pciutil
    if (path_length <= 5) continue;
    int kNumHops = std::min(absl::GetFlag(FLAGS_num_hops), MAX_HOPS);
    char *pci_addr = nullptr;
    for (int i = 0; i < kNumHops; i++) {
      char *slash = strrchr(if_sysfs_realpath, '/');
      /* First delimiter gives us the pci address*/
      if (i == 0) pci_addr = slash + 1;
      *slash = '\0';
    }
    uint16_t temp_domain, temp_bus, temp_device, temp_function;
    /* Not a valid PCI address */
    LOG(INFO) << "PCI addr for net if " << head->ifa_name << ": "
              << std::string(pci_addr);
    if (parse_pci_addr(pci_addr, &temp_domain, &temp_bus, &temp_device,
                       &temp_function))
      continue;
    netdev_to_pci.emplace(head->ifa_name, pci_addr);
    LOG(INFO) << "Root dir: " << if_sysfs_realpath;
    std::vector<std::string> candidates;
    list_vendor_devices(if_sysfs_realpath, &candidates, "0x10de");
    for (auto &candidate : candidates) {
      LOG(INFO) << "Potential candidate: " << candidate;
      pci_to_netdev.emplace(absl::AsciiStrToLower(candidate), head->ifa_name);
    }
  } while ((head = head->ifa_next) != nullptr);
  // Get PCI addrs for CUDA devices and find the closest NIC.
  int num_cuda_device = 0;
  CUDA_ASSERT_SUCCESS(hipGetDeviceCount(&num_cuda_device));
  for (int i = 0; i < num_cuda_device; i++) {
    char gpu_pci_addr[MAX_PCI_ADDR_LEN];
    CUDA_ASSERT_SUCCESS(
        hipDeviceGetPCIBusId(gpu_pci_addr, MAX_PCI_ADDR_LEN, i));
    for (int i = 0; i < MAX_PCI_ADDR_LEN; i++) {
      gpu_pci_addr[i] = tolower(gpu_pci_addr[i]);
    }
    if (!pci_to_netdev.contains(gpu_pci_addr)) {
      LOG(ERROR) << "Cannot find corresponding GPU NIC for GPU " << gpu_pci_addr
                 << ".";
      continue;
    }
    auto &netdev_name = pci_to_netdev[gpu_pci_addr];
    if (!netdev_to_pci.contains(netdev_name)) {
      LOG(ERROR) << "Net dev " << netdev_name << " is not discovered before.";
      continue;
    }
    LOG(INFO) << "Corresponding PCI NIC for GPU PCI addr " << gpu_pci_addr
              << " is " << netdev_name;

    netdev_to_gpu_pcis[netdev_name].push_back(gpu_pci_addr);
  }

  for (const auto &[netdev_name, gpu_pci_addrs] : netdev_to_gpu_pcis) {
    GpuRxqConfiguration configuration;
    int queue_start = kRssSetSize;
    int queue_count = kTcpdQueueCount / gpu_pci_addrs.size();
    CHECK(queue_count > 0);
    for (const auto &gpu_pci_addr : gpu_pci_addrs) {
      GpuInfo *gpu_info = configuration.add_gpu_infos();
      gpu_info->set_gpu_pci_addr(gpu_pci_addr);
      for (int i = queue_start; i < queue_start + queue_count; ++i) {
        gpu_info->add_queue_ids(i);
      }
      queue_start += queue_count;
    }
    configuration.set_nic_pci_addr(netdev_to_pci[netdev_name]);
    configuration.set_ifname(netdev_name);
    *(config_list.add_gpu_rxq_configs()) = std::move(configuration);
  }

  freeifaddrs(all_ifs);
  config_list.set_tcpd_queue_size(kTcpdQueueCount);
  config_list.set_rss_set_size(kRssSetSize);
  return config_list;
}
}  // namespace gpudirect_tcpxd
